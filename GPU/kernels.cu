#include "hip/hip_runtime.h"
#include "kernels.cuh"
#define THREADS_PER_BLOCK 256
#define BLOCK_COUNT(SIZE) ((SIZE) / THREADS_PER_BLOCK + (((SIZE) % THREADS_PER_BLOCK) ? 1 : 0))

__global__ void
k_vector_add(float* A, const float* B, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numElements)
		A[i] = A[i] + B[i];
}

/// Add two vectors
void vector_add(float* A, const float* B, int numElements) {
	k_vector_add << <1, numElements >> > (A, B, numElements);
}

__device__ void
d_dot(const float* v1, const float* v2, float* out, int size)
{
	__shared__ float cache[THREADS_PER_BLOCK];
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	cache[threadIdx.x] = 0.f;
	while (i < size) {
		cache[threadIdx.x] += v1[i] * v2[i];
		i += gridDim.x * blockDim.x;
	}
	__syncthreads();
	i = THREADS_PER_BLOCK / 2;
	while (i > 0) {
		if (threadIdx.x < i) cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;
	}
	if (threadIdx.x == 0) atomicAdd(out, cache[0]);
}

__global__ void
k_dot(const float* V1, const float* V2, float* V3, int size)
{
	d_dot(V1, V2, V3, size);
}

__global__ void
k_update(float learn_rate, float* expected, float* data, float* bias, float* weights, int size, float* result)
{
	d_dot(data, weights, result, size);
	if (blockIdx.x == 0 && threadIdx.x == 0)
	{
		*result = (*bias + *result) > 0 ? 1.f : -1.f;
		*result = learn_rate * (*expected - *result);
	}
}

/// Compute dot product of two vectors
float* dot(float* a, float* b, int size) {
	float* c;
	gpuErrchk(hipMalloc(&c, 1 * sizeof(float)));
	int bc = BLOCK_COUNT(size);
	k_dot << <bc, THREADS_PER_BLOCK >> > (a, b, c, size);
	return c;
}

/// Compute udpate value for training
float* update(float learn_rate, float* expected, float* data, float* bias, float* weights, int size)
{
	float* result;
	gpuErrchk(hipMalloc(&result, 1 * sizeof(float)));
	int bc = BLOCK_COUNT(size);
	k_update << <bc, THREADS_PER_BLOCK >> > (learn_rate, expected, data, bias, weights, size, result);
	return result;
}

__global__ void
k_scale(float* scaler, float* vector, float* result, int size) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	float s = *scaler;
	if (index < size) {
		result[index] += vector[index] * s;
	}
}

/// Scale vector with a given scaler and save to result
void scale(float* scaler, float* vector, float* result, int size) {
	k_scale << <1, size >> > (scaler, vector, result, size);
}

__global__ void
k_classify(float* data, float* weights, float* bias, float* result, int length, int size)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < length) {
		float* current = data + index * size;
		float prob = *bias;
		for (int i = 0; i < size; i++)
		{
			prob += current[i] * weights[i];
		}
		result[index] = prob > 0 ? 1.f : -1.f;
	}
}

float* classify(float* data, float* weights, float* bias, int length, int size)
{
	float* result;
	gpuErrchk(hipMalloc(&result, length * sizeof(float)));
	int bc = BLOCK_COUNT(length);
	k_classify << < bc, THREADS_PER_BLOCK >> > (data, weights, bias, result, length, size);
	return result;
}